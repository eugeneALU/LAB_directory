// GPU version of Matrix addition


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

const int N = 1024;
const int blocksize = (N>32)?32:N;           //MAX threads per block = 1024 Sqrt(1024)=32

__global__
void add_matrix(float *a, float *b, float *c, int N, int gridsize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int id = idy*gridsize*blockDim.x + idx;
  if(id < N*N){
    c[id] = a[id] + b[id];
  }
}

int main()
{
  float *a = new float[N*N];
  float *b = new float[N*N];
  float *c = new float[N*N];
  float *a_g;
  float *b_g;
  float *c_g;
  float t;    //excution time in ms
  int gridsize;
  gridsize =(int)ceil((double)N/blocksize); //handle for the situation that N%blocksize != 0

  size_t size =  N*N*sizeof(float);
  hipEvent_t start, end;
  /*
  cudaDeviceProp prop;
  cudaGetDeviceProperties(&prop, 0);
  printf("Name: %s\n",prop.name);
  printf("MAX Threads per block: %d\n", prop.maxThreadsPerBlock);
  printf("MAX Grid: [%d %d %d]\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
  printf("MAX shared Mem per block: %lu\n", prop.sharedMemPerBlock);
  //more property can be find : https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html#group__CUDART__DEVICE_1g1bf9d625a931d657e08db2b4391170f0
  */
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);
  hipEventRecord(end, 0);

  dim3 dimBlock(blocksize, blocksize);
  dim3 dimGrid(gridsize, gridsize);
  hipMalloc((void**)&a_g, size);
  hipMalloc((void**)&b_g, size);
  hipMalloc((void**)&c_g, size);

  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      a[i+j*N] = 10 + i;
      b[i+j*N] = (float)j / N;
    }
  }
  hipMemcpy(a_g, a, size, hipMemcpyHostToDevice);
  hipMemcpy(b_g, b, size, hipMemcpyHostToDevice);

  hipEventSynchronize(start);
  add_matrix<<<dimGrid, dimBlock>>>(a_g, b_g, c_g, N, gridsize);
  hipDeviceSynchronize();
  hipEventSynchronize(end);
  hipEventElapsedTime(&t, start, end);

  hipMemcpy(c, c_g, size, hipMemcpyDeviceToHost);
    
    int i;
    FILE *f = fopen("gpu.txt", "wb");
    for (i = 0; i < N*N; i++) {
      fprintf(f, "%f\n", c[i]);
    }
    fclose(f);   
/*
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      printf("%0.2f ", c[i+j*N]);
    }
    printf("\n");
  }*/
  delete[] a;
  delete[] b;
  delete[] c;
  hipFree(a_g);
  hipFree(b_g);
  hipFree(c_g);

  printf("Cost %0.8f miliseconds\n", t);
  return EXIT_SUCCESS;
}
