#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10
#define BLOCKSIZE 32

// filter in x direction (imagesizex)
__global__ void filter(unsigned char *image, unsigned char *out, unsigned int imagesizex, unsigned int imagesizey,
                       const int kernelsize, const int intervalx, const int intervaly)
{
    int globalx = blockIdx.x * blockDim.x + threadIdx.x;
    int globaly = blockIdx.y * blockDim.y + threadIdx.y;
    int localx = threadIdx.x;
    int localy = threadIdx.y;

    //SWAP X and Y direction of global ID
    if (intervalx == imagesizex){
        globaly = blockIdx.x * blockDim.x + threadIdx.x;
        globalx = blockIdx.y * blockDim.y + threadIdx.y;
        localy = threadIdx.x;
        localx = threadIdx.y;
         //also need to swap image size incase non square input image, so input parameter has been swap
    }
    __shared__ unsigned char local_mem[(BLOCKSIZE+2*maxKernelSizeX)*3*BLOCKSIZE];

    int length = BLOCKSIZE+2*maxKernelSizeX;
    int offset = kernelsize;
    int xx = min(globalx, imagesizex-1);  //clamping
    int yy = min(globaly, imagesizey-1);
    int idx, d;

    local_mem[(localy*length+localx+offset)*3+0] = image[((yy)*intervaly+(xx)*intervalx)*3+0];
    local_mem[(localy*length+localx+offset)*3+1] = image[((yy)*intervaly+(xx)*intervalx)*3+1];
    local_mem[(localy*length+localx+offset)*3+2] = image[((yy)*intervaly+(xx)*intervalx)*3+2];

    int x;
    if(localx < kernelsize){
        x = max(xx-kernelsize,0);
        idx = yy*intervaly+x*intervalx;
        //idx = max(globaly*imagesizex+globalx - kernelsize*interval, 0);

        local_mem[(localy*length+localx)*3+0] = image[(idx)*3+0];
        local_mem[(localy*length+localx)*3+1] = image[(idx)*3+1];
        local_mem[(localy*length+localx)*3+2] = image[(idx)*3+2];
    }
    if (localx > BLOCKSIZE-1 - kernelsize){
        x = min(xx+kernelsize, imagesizex-1);
        idx = yy*intervaly+x*intervalx;
        //idx = min(globaly*imagesizex+globalx + kernelsize*interval, imagesizey*imagesizex);

        local_mem[(localy*length+localx+2*offset)*3+0] = image[(idx)*3+0];
        local_mem[(localy*length+localx+2*offset)*3+1] = image[(idx)*3+1];
        local_mem[(localy*length+localx+2*offset)*3+2] = image[(idx)*3+2];
    }
    __syncthreads();

    float weight[5] ={1.0/16,4.0/16,6.0/16,4.0/16,1.0/16};
    int i = 0;
    float sumx, sumy, sumz;

	// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(d=-kernelsize;d<=kernelsize;d++)
	{
        //x = min(max(localx+d,0),BLOCKSIZE+kernelsize);
        //idx = localy*length+x+offset;
        idx = localy*length+localx+offset + d;

		sumx += local_mem[idx*3+0]*weight[i];
		sumy += local_mem[idx*3+1]*weight[i];
		sumz += local_mem[idx*3+2]*weight[i];

        i++;
	}

    out[((yy)*intervaly+(xx)*intervalx)*3+0] =  sumx;
    out[((yy)*intervaly+(xx)*intervalx)*3+1] =  sumy;
    out[((yy)*intervaly+(xx)*intervalx)*3+2] =  sumz;
}

// Global variables for image data
unsigned char *image, *pixels, *dev_bitmap, *dev_input, *intermediate;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
  double t;
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}

    pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
    hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
    hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
    hipMalloc( (void**)&intermediate, imagesizex*imagesizey*3);
    hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

	dim3 grid(ceil(float(imagesizex)/(BLOCKSIZE)),ceil(float(imagesizey)/(BLOCKSIZE)));
	dim3 block(BLOCKSIZE,BLOCKSIZE); // change to blocksize = 32*32
    ResetMilli();
    // row wise, interval between each target = 1
    filter<<<grid,block>>>(dev_input, intermediate, imagesizex, imagesizey, kernelsizex, 1, imagesizex);
    hipDeviceSynchronize();
    // col wise, interval between each target = blocksize (=32)
    filter<<<grid,block>>>(intermediate, dev_bitmap, imagesizey, imagesizex, kernelsizey, imagesizex, 1);
    hipDeviceSynchronize();
    t = GetSeconds();
    printf("COST %lf seconds\n", t);
    //	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
      printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
    hipFree( dev_bitmap );
    hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(2, 2); // 1 * 5 gausian kernel

	// You can save the result to a file like this:
	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
