#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10
#define BLOCKSIZE 32


__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
	// map from blockIdx to pixel position
	int globalx = blockIdx.x * blockDim.x + threadIdx.x;
	int globaly = blockIdx.y * blockDim.y + threadIdx.y;
	int localx = threadIdx.x;
	int localy = threadIdx.y;

	int dy, dx;
	unsigned int sumx, sumy, sumz;

	__shared__ unsigned char local_mem[(BLOCKSIZE+2*maxKernelSizeX)*3][BLOCKSIZE+2*maxKernelSizeY];
	int yy = min(max(globaly, 0), imagesizey-1);
	int xx = min(max(globalx, 0), imagesizex-1);
	// copy core section
	local_mem[(kernelsizex+localx)*3+0][kernelsizey+localy] = image[((yy)*imagesizex+(xx))*3+0];
	local_mem[(kernelsizex+localx)*3+1][kernelsizey+localy] = image[((yy)*imagesizex+(xx))*3+1];
	local_mem[(kernelsizex+localx)*3+2][kernelsizey+localy] = image[((yy)*imagesizex+(xx))*3+2];

     // handle up and down edge
    yy = min(max(globaly, 0), imagesizey-1);
	if (localx < kernelsizex)
	{
		xx = max(globalx-kernelsizex, 0); 	//if(globalx-kernalsizex < 0)
		local_mem[(localx)*3 + 0][kernelsizey+localy] = image[(yy*imagesizex + xx)*3+0];
		local_mem[(localx)*3 + 1][kernelsizey+localy] = image[(yy*imagesizex + xx)*3+1];
		local_mem[(localx)*3 + 2][kernelsizey+localy] = image[(yy*imagesizex + xx)*3+2];
	}
	else if (localx > BLOCKSIZE-1 -kernelsizex)
	{
		xx = min(globalx+kernelsizex, imagesizex-1); 	//if(globalx+kernalsizex > imagesizex)
		local_mem[(localx+2*kernelsizex)*3 + 0][kernelsizey+localy] = image[(yy*imagesizex + xx)*3+0];
		local_mem[(localx+2*kernelsizex)*3 + 1][kernelsizey+localy] = image[(yy*imagesizex + xx)*3+1];
		local_mem[(localx+2*kernelsizex)*3 + 2][kernelsizey+localy] = image[(yy*imagesizex + xx)*3+2];
	}

    //handle left and right edge
    xx = min(max(globalx, 0), imagesizex-1);
	if (localy < kernelsizey)
	{
		yy = max(globaly-kernelsizey, 0); 	//if(globaly-kernalsizey < 0)
		local_mem[(kernelsizex+localx)*3+0][localy] = image[(yy*imagesizex + xx)*3+0];
		local_mem[(kernelsizex+localx)*3+1][localy] = image[(yy*imagesizex + xx)*3+1];
		local_mem[(kernelsizex+localx)*3+2][localy] = image[(yy*imagesizex + xx)*3+2];
	}
	else if (localy > BLOCKSIZE-1 -kernelsizey)
	{
		yy = min(globaly+kernelsizey, imagesizey-1); 	//if(globaly+kernalsizey > 0)
		local_mem[(kernelsizex+localx)*3+0][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+0];
		local_mem[(kernelsizex+localx)*3+1][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+1];
		local_mem[(kernelsizex+localx)*3+2][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+2];
	}

    //handle corner 
    if (localx < kernelsizex && localy < kernelsizey)
    {
        xx = max(globalx-kernelsizex, 0); 	//if(globalx-kernalsizex < 0)
        yy = max(globaly-kernelsizey, 0); 	//if(globaly-kernalsizey < 0)
        local_mem[localx*3 + 0][localy] = image[(yy*imagesizex + xx)*3+0];
        local_mem[localx*3 + 1][localy] = image[(yy*imagesizex + xx)*3+1];
        local_mem[localx*3 + 2][localy] = image[(yy*imagesizex + xx)*3+2];
    }
    else if (localx > BLOCKSIZE-1 - kernelsizex && localy > BLOCKSIZE-1 - kernelsizey)
    {
        xx = min(globalx+kernelsizex, imagesizex-1); 	//if(globalx+kernalsizex > imagesizex)
        yy = min(globaly+kernelsizey, imagesizey-1); 	//if(globaly+kernalsizey > imagesizey)
        local_mem[(localx+2*kernelsizex)*3 + 0][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+0];
        local_mem[(localx+2*kernelsizex)*3 + 1][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+1];
        local_mem[(localx+2*kernelsizex)*3 + 2][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+2];
    }
    else if (localx > BLOCKSIZE-1 - kernelsizex &&  localy < kernelsizey)
    {
        xx = min(globalx+kernelsizex, imagesizex-1); 	//if(globalx+kernalsizex > imagesizex)
        yy = max(globaly-kernelsizey, 0); 	            //if(globaly-kernalsizey < 0)
        local_mem[(localx+2*kernelsizex)*3 + 0][localy] = image[(yy*imagesizex + xx)*3+0];
        local_mem[(localx+2*kernelsizex)*3 + 1][localy] = image[(yy*imagesizex + xx)*3+1];
        local_mem[(localx+2*kernelsizex)*3 + 2][localy] = image[(yy*imagesizex + xx)*3+2];
    }
    else if (localx < kernelsizex && localy > BLOCKSIZE-1 - kernelsizey)
    {
        xx = max(globalx-kernelsizex, 0); 	            //if(globalx-kernalsizex < 0)
        yy = min(globaly+kernelsizey, imagesizey-1); 	//if(globaly+kernalsizey > imagesizey)
        local_mem[(localx)*3 + 0][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+0];
        local_mem[(localx)*3 + 1][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+1];
        local_mem[(localx)*3 + 2][localy+2*kernelsizey] = image[(yy*imagesizex + xx)*3+2];
    }
	__syncthreads();

	int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!

	if (globalx < imagesizex && globaly < imagesizey) // If inside image
	{
		// Filter kernel (simple box filter)
		sumx=0;sumy=0;sumz=0;
		for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		{
			for(dx=-kernelsizex;dx<=kernelsizex;dx++)
			{
				// Use max and min to avoid branching!
				yy = min(max(localy+kernelsizey +dy, 0), BLOCKSIZE+2*kernelsizey-1);
				xx = min(max(localx+kernelsizex +dx, 0), BLOCKSIZE+2*kernelsizex-1);

				sumx += local_mem[(xx)*3+0][yy];
				sumy += local_mem[(xx)*3+1][yy];
				sumz += local_mem[(xx)*3+2][yy];
			}
		}
		out[((globaly)*imagesizex+(globalx))*3+0] = sumx/divby;
		out[((globaly)*imagesizex+(globalx))*3+1] = sumy/divby;
		out[((globaly)*imagesizex+(globalx))*3+2] = sumz/divby;
        /*out[((globaly)*imagesizex+(globalx))*3+0] = local_mem[(kernelsizex+localx)*3+0][kernelsizey+localy];
		out[((globaly)*imagesizex+(globalx))*3+1] = local_mem[(kernelsizex+localx)*3+1][kernelsizey+localy];
		out[((globaly)*imagesizex+(globalx))*3+2] = local_mem[(kernelsizex+localx)*3+2][kernelsizey+localy];*/
	}
}

// Global variables for image data

unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
    double t;
    if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
    {
        printf("Kernel size out of bounds!\n");
        return;
    }

    pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
    hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
    hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
    hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);
    dim3 grid(ceil(float(imagesizex)/(BLOCKSIZE)),ceil(float(imagesizey)/(BLOCKSIZE)));
    dim3 block(BLOCKSIZE,BLOCKSIZE);
    ResetMilli();
    filter<<<grid,block>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // change to blocksize = 32*32
    hipDeviceSynchronize();
    t = GetSeconds();
    printf("COST %lf seconds\n", t);
    //	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
    hipFree( dev_bitmap );
    hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(7, 7);

	// You can save the result to a file like this:
	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
