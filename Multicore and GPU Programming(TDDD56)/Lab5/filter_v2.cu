#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10
#define BLOCKSIZE 32

__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
    int localx = threadIdx.x;
    int localy = threadIdx.y;

    int dy, dx;
    unsigned int sumx, sumy, sumz;

    __shared__ unsigned char local_mem[BLOCKSIZE*3][BLOCKSIZE];
    int startx = (BLOCKSIZE-2*kernelsizex)*blockIdx.x;  //block start index
    int starty = (BLOCKSIZE-2*kernelsizey)*blockIdx.y;
    int realx = localx - kernelsizex + startx;          //real image idx that local idx corresponding to
    int realy = localy - kernelsizey + starty;  
    int xx = min(max(realx, 0), imagesizex-1);          //clamping
    int yy = min(max(realy, 0), imagesizey-1);

    local_mem[3*localx+0][localy] = image[((yy)*imagesizex+(xx))*3+0];
    local_mem[3*localx+1][localy] = image[((yy)*imagesizex+(xx))*3+1];
    local_mem[3*localx+2][localy] = image[((yy)*imagesizex+(xx))*3+2];
    __syncthreads();

	int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!
    int x,y;
	// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
	{
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)
		{
            x = min(max(localx+dx, 0), BLOCKSIZE-1);
            y = min(max(localy+dy, 0), BLOCKSIZE-1);

            sumx += local_mem[x*3+0][y];
            sumy += local_mem[x*3+1][y];
            sumz += local_mem[x*3+2][y];
		}
	}
    if (realx >= 0 && realx <= imagesizex - 1){
        if (realy >= 0 && realy <= imagesizey - 1){
            out[((yy)*imagesizex+(xx))*3+0] = sumx/divby;
            out[((yy)*imagesizex+(xx))*3+1] = sumy/divby;
            out[((yy)*imagesizex+(xx))*3+2] = sumz/divby;
            /*out[((yy)*imagesizex+(xx))*3+0] =local_mem[3*localx+0][localy];
            out[((yy)*imagesizex+(xx))*3+1] =local_mem[3*localx+1][localy];
            out[((yy)*imagesizex+(xx))*3+2] =local_mem[3*localx+2][localy];*/
        }
    }
}

// Global variables for image data
unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
    double t;
    if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
    {
	    printf("Kernel size out of bounds!\n");
	    return;
    }

    pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
    hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
    hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
    hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);
    printf("GRIDSIZE:%f\n", ceil(float(imagesizex)/(32-2*kernelsizex)));
    dim3 grid(ceil(float(imagesizex)/(BLOCKSIZE-2*kernelsizex)),ceil(float(imagesizey)/(BLOCKSIZE-2*kernelsizey)));
    dim3 block(BLOCKSIZE,BLOCKSIZE);
    ResetMilli();
    filter<<<grid,block>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // change to blocksize = 32*32
    hipDeviceSynchronize();
    t = GetSeconds();
    printf("COST %lf seconds\n", t);
    //	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
    hipFree( dev_bitmap );
    hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(7, 7);

	// You can save the result to a file like this:
	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
